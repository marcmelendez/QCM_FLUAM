// Filename: freeBoundariesRK2GPU.cu
//
// Copyright (c) 2010-2016, Florencio Balboa Usabiaga
//
// This file is part of Fluam
//
// Fluam is free software: you can redistribute it and/or modify
// it under the terms of the GNU General Public License as published by
// the Free Software Foundation, either version 3 of the License, or
// (at your option) any later version.
//
// Fluam is distributed in the hope that it will be useful,
// but WITHOUT ANY WARRANTY; without even the implied warranty of
// MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the
// GNU General Public License for more details.
//
// You should have received a copy of the GNU General Public License
// along with Fluam. If not, see <http://www.gnu.org/licenses/>.


bool freeBoundariesRK2BigSystemGPU(){

  cutilSafeCall(hipUnbindTexture(texrxboundaryGPU));    
  cutilSafeCall(hipUnbindTexture(texryboundaryGPU));    
  cutilSafeCall(hipUnbindTexture(texrzboundaryGPU));

  cutilSafeCall(hipFree(rxboundaryGPU));
  cutilSafeCall(hipFree(ryboundaryGPU));
  cutilSafeCall(hipFree(rzboundaryGPU));
  cutilSafeCall(hipFree(rxboundaryPredictionGPU));
  cutilSafeCall(hipFree(ryboundaryPredictionGPU));
  cutilSafeCall(hipFree(rzboundaryPredictionGPU));
  cutilSafeCall(hipFree(vxboundaryGPU));
  cutilSafeCall(hipFree(vyboundaryGPU));
  cutilSafeCall(hipFree(vzboundaryGPU));
  cutilSafeCall(hipFree(vxboundaryPredictionGPU));
  cutilSafeCall(hipFree(vyboundaryPredictionGPU));
  cutilSafeCall(hipFree(vzboundaryPredictionGPU));
  cutilSafeCall(hipFree(fxboundaryGPU));
  cutilSafeCall(hipFree(fyboundaryGPU));
  cutilSafeCall(hipFree(fzboundaryGPU));


  if(setparticles){
    cutilSafeCall(hipFree(countPartInCellNonBonded));
    cutilSafeCall(hipFree(partInCellNonBonded));

    cutilSafeCall(hipFree(neighbor0GPU));
    cutilSafeCall(hipFree(neighbor1GPU));
    cutilSafeCall(hipFree(neighbor2GPU));
    cutilSafeCall(hipFree(neighbor3GPU));
    cutilSafeCall(hipFree(neighbor4GPU));
    cutilSafeCall(hipFree(neighbor5GPU));
    cutilSafeCall(hipFree(neighborpxpyGPU));
    cutilSafeCall(hipFree(neighborpxmyGPU));
    cutilSafeCall(hipFree(neighborpxpzGPU));
    cutilSafeCall(hipFree(neighborpxmzGPU));
    cutilSafeCall(hipFree(neighbormxpyGPU));
    cutilSafeCall(hipFree(neighbormxmyGPU));
    cutilSafeCall(hipFree(neighbormxpzGPU));
    cutilSafeCall(hipFree(neighbormxmzGPU));
    cutilSafeCall(hipFree(neighborpypzGPU));
    cutilSafeCall(hipFree(neighborpymzGPU));
    cutilSafeCall(hipFree(neighbormypzGPU));
    cutilSafeCall(hipFree(neighbormymzGPU));
    cutilSafeCall(hipFree(neighborpxpypzGPU));
    cutilSafeCall(hipFree(neighborpxpymzGPU));
    cutilSafeCall(hipFree(neighborpxmypzGPU));
    cutilSafeCall(hipFree(neighborpxmymzGPU));
    cutilSafeCall(hipFree(neighbormxpypzGPU));
    cutilSafeCall(hipFree(neighbormxpymzGPU));
    cutilSafeCall(hipFree(neighbormxmypzGPU));
    cutilSafeCall(hipFree(neighbormxmymzGPU));
    cutilSafeCall(hipFree(pNeighbors));
  }

  freeErrorArray();
  cutilSafeCall(hipFree(pc));
  freeDelta();

  if(setparticles){
    cutilSafeCall(hipUnbindTexture(texforceNonBonded1));
    cutilSafeCall(hipFreeArray(forceNonBonded1));
  }


  cout << "FREE BOUNDARIES GPU :           DONE" << endl; 

  return 1;
}
