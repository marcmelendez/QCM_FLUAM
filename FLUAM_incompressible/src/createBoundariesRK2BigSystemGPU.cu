// Filename: createBoundariesRK2GPU.cu
//
// Copyright (c) 2010-2016, Florencio Balboa Usabiaga
//
// This file is part of Fluam
//
// Fluam is free software: you can redistribute it and/or modify
// it under the terms of the GNU General Public License as published by
// the Free Software Foundation, either version 3 of the License, or
// (at your option) any later version.
//
// Fluam is distributed in the hope that it will be useful,
// but WITHOUT ANY WARRANTY; without even the implied warranty of
// MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the
// GNU General Public License for more details.
//
// You should have received a copy of the GNU General Public License
// along with Fluam. If not, see <http://www.gnu.org/licenses/>.


bool createBoundariesRK2BigSystemGPU(){

  if(setparticles==0) np=0;
  
  hipMemcpyToSymbol(HIP_SYMBOL(nboundaryGPU),&nboundary,sizeof(int));
  hipMemcpyToSymbol(HIP_SYMBOL(npGPU),&np,sizeof(int));
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(maxNumberPartInCellGPU),&maxNumberPartInCell,sizeof(int)));
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(maxNumberPartInCellNonBondedGPU),&maxNumberPartInCellNonBonded,sizeof(int)));
  //*!R new parameter
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(confinementZGPU),&confinementZ,sizeof(bool)));
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(confinementZKGPU),&confinementZK,sizeof(double)));

  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(particlesWallGPU),&particlesWall,sizeof(bool)));
  hipMemcpyToSymbol(HIP_SYMBOL(computeNonBondedForcesGPU),&computeNonBondedForces,sizeof(bool));

  //Create boundaries and particles variables
  cutilSafeCall(hipMalloc((void**)&rxboundaryGPU,(nboundary+np)*sizeof(double)));
  cutilSafeCall(hipMalloc((void**)&ryboundaryGPU,(nboundary+np)*sizeof(double)));
  cutilSafeCall(hipMalloc((void**)&rzboundaryGPU,(nboundary+np)*sizeof(double)));
  cutilSafeCall(hipMalloc((void**)&rxboundaryPredictionGPU,(nboundary+np)*sizeof(double)));
  cutilSafeCall(hipMalloc((void**)&ryboundaryPredictionGPU,(nboundary+np)*sizeof(double)));
  cutilSafeCall(hipMalloc((void**)&rzboundaryPredictionGPU,(nboundary+np)*sizeof(double)));
  cutilSafeCall(hipMalloc((void**)&vxboundaryGPU,(nboundary+np)*sizeof(double)));
  cutilSafeCall(hipMalloc((void**)&vyboundaryGPU,(nboundary+np)*sizeof(double)));
  cutilSafeCall(hipMalloc((void**)&vzboundaryGPU,(nboundary+np)*sizeof(double)));
  cutilSafeCall(hipMalloc((void**)&vxboundaryPredictionGPU,(nboundary+np)*sizeof(double)));
  cutilSafeCall(hipMalloc((void**)&vyboundaryPredictionGPU,(nboundary+np)*sizeof(double)));
  cutilSafeCall(hipMalloc((void**)&vzboundaryPredictionGPU,(nboundary+np)*sizeof(double)));
  cutilSafeCall(hipMalloc((void**)&fxboundaryGPU,27*(nboundary+np)*sizeof(double)));
  cutilSafeCall(hipMalloc((void**)&fyboundaryGPU,27*(nboundary+np)*sizeof(double)));
  cutilSafeCall(hipMalloc((void**)&fzboundaryGPU,27*(nboundary+np)*sizeof(double)));

  //Initialize boundaries variables
  cutilSafeCall(hipMemcpy(rxboundaryGPU,rxboundary,nboundary*sizeof(double),hipMemcpyHostToDevice));
  cutilSafeCall(hipMemcpy(ryboundaryGPU,ryboundary,nboundary*sizeof(double),hipMemcpyHostToDevice));
  cutilSafeCall(hipMemcpy(rzboundaryGPU,rzboundary,nboundary*sizeof(double),hipMemcpyHostToDevice));
  cutilSafeCall(hipMemcpy(vxboundaryGPU,vxboundary,nboundary*sizeof(double),hipMemcpyHostToDevice));
  cutilSafeCall(hipMemcpy(vyboundaryGPU,vyboundary,nboundary*sizeof(double),hipMemcpyHostToDevice));
  cutilSafeCall(hipMemcpy(vzboundaryGPU,vzboundary,nboundary*sizeof(double),hipMemcpyHostToDevice));

  //Initialize particles variables
  cutilSafeCall(hipMemcpy(&rxboundaryGPU[nboundary],rxParticle,np*sizeof(double),hipMemcpyHostToDevice));
  cutilSafeCall(hipMemcpy(&ryboundaryGPU[nboundary],ryParticle,np*sizeof(double),hipMemcpyHostToDevice));
  cutilSafeCall(hipMemcpy(&rzboundaryGPU[nboundary],rzParticle,np*sizeof(double),hipMemcpyHostToDevice));
  cutilSafeCall(hipMemcpy(&vxboundaryGPU[nboundary],vxParticle,np*sizeof(double),hipMemcpyHostToDevice));
  cutilSafeCall(hipMemcpy(&vyboundaryGPU[nboundary],vyParticle,np*sizeof(double),hipMemcpyHostToDevice));
  cutilSafeCall(hipMemcpy(&vzboundaryGPU[nboundary],vzParticle,np*sizeof(double),hipMemcpyHostToDevice));
  
  //Copy some constants
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(volumeboundaryconstGPU),&volumeboundaryconst,sizeof(double)));
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(massParticleGPU),&mass,sizeof(double)));
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(volumeParticleGPU),&volumeParticle,sizeof(double)));
 
              
  cutilSafeCall(hipMalloc((void**)&countparticlesincellX,ncells*sizeof(int)));
  cutilSafeCall(hipMalloc((void**)&countparticlesincellY,ncells*sizeof(int)));
  cutilSafeCall(hipMalloc((void**)&countparticlesincellZ,ncells*sizeof(int)));

  int *aux;
  aux = new int [ncells];
  for(int i=0;i<ncells;i++) aux[i] = 0;
  hipMemcpy(countparticlesincellX,aux,ncells*sizeof(int),hipMemcpyHostToDevice);
  hipMemcpy(countparticlesincellY,aux,ncells*sizeof(int),hipMemcpyHostToDevice);
  hipMemcpy(countparticlesincellZ,aux,ncells*sizeof(int),hipMemcpyHostToDevice);
  delete[] aux;

  hipMalloc((void**)&partincellX,maxNumberPartInCell*ncells*sizeof(int));
  hipMalloc((void**)&partincellY,maxNumberPartInCell*ncells*sizeof(int));
  hipMalloc((void**)&partincellZ,maxNumberPartInCell*ncells*sizeof(int));
  

  //texrxboundaryGPU
  texrxboundaryGPU.normalized = false;
  texrxboundaryGPU.addressMode[0] = hipAddressModeClamp;//Wrap and Clamp
  texrxboundaryGPU.filterMode = hipFilterModePoint;//hipFilterModeLinear and hipFilterModePoint
  cutilSafeCall( hipBindTexture(0,texrxboundaryGPU,rxboundaryGPU,(nboundary+np)*sizeof(double)));
  //texryboundaryGPU
  texryboundaryGPU.normalized = false;
  texryboundaryGPU.addressMode[0] = hipAddressModeClamp;//Wrap and Clamp
  texryboundaryGPU.filterMode = hipFilterModePoint;//hipFilterModeLinear and hipFilterModePoint
  cutilSafeCall( hipBindTexture(0,texryboundaryGPU,ryboundaryGPU,(nboundary+np)*sizeof(double)));
  //texrzboundaryGPU
  texrzboundaryGPU.normalized = false;
  texrzboundaryGPU.addressMode[0] = hipAddressModeClamp;//Wrap and Clamp
  texrzboundaryGPU.filterMode = hipFilterModePoint;//hipFilterModeLinear and hipFilterModePoint
  cutilSafeCall( hipBindTexture(0,texrzboundaryGPU,rzboundaryGPU,(nboundary+np)*sizeof(double)));

  if(setparticles){
    int mxPart = int(lx/cutoff);
    if(mxPart < 3) mxPart = 3;
    int myPart = int(ly/cutoff);
    if(myPart < 3) myPart = 3;
    int mzPart = int(lz/cutoff);
    if(mzPart < 3) mzPart = 3;
    numNeighbors = mxPart * myPart * mzPart;

    cutilSafeCall(hipMalloc((void**)&countPartInCellNonBonded,numNeighbors*sizeof(int)));
    cutilSafeCall(hipMalloc((void**)&partInCellNonBonded,
			     maxNumberPartInCellNonBonded*numNeighbors*sizeof(int)));

    cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(mxNeighborsGPU),&mxPart,sizeof(int)));
    cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(myNeighborsGPU),&myPart,sizeof(int)));
    cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(mzNeighborsGPU),&mzPart,sizeof(int)));
    cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(mNeighborsGPU),&numNeighbors,sizeof(int)));
    
    cutilSafeCall(hipMalloc((void**)&neighbor0GPU,numNeighbors*sizeof(int)));
    cutilSafeCall(hipMalloc((void**)&neighbor1GPU,numNeighbors*sizeof(int)));
    cutilSafeCall(hipMalloc((void**)&neighbor2GPU,numNeighbors*sizeof(int)));
    cutilSafeCall(hipMalloc((void**)&neighbor3GPU,numNeighbors*sizeof(int)));
    cutilSafeCall(hipMalloc((void**)&neighbor4GPU,numNeighbors*sizeof(int)));
    cutilSafeCall(hipMalloc((void**)&neighbor5GPU,numNeighbors*sizeof(int)));
    cutilSafeCall(hipMalloc((void**)&neighborpxpyGPU,numNeighbors*sizeof(int)));
    cutilSafeCall(hipMalloc((void**)&neighborpxmyGPU,numNeighbors*sizeof(int)));
    cutilSafeCall(hipMalloc((void**)&neighborpxpzGPU,numNeighbors*sizeof(int)));
    cutilSafeCall(hipMalloc((void**)&neighborpxmzGPU,numNeighbors*sizeof(int)));
    cutilSafeCall(hipMalloc((void**)&neighbormxpyGPU,numNeighbors*sizeof(int)));
    cutilSafeCall(hipMalloc((void**)&neighbormxmyGPU,numNeighbors*sizeof(int)));
    cutilSafeCall(hipMalloc((void**)&neighbormxpzGPU,numNeighbors*sizeof(int)));
    cutilSafeCall(hipMalloc((void**)&neighbormxmzGPU,numNeighbors*sizeof(int)));
    cutilSafeCall(hipMalloc((void**)&neighborpypzGPU,numNeighbors*sizeof(int)));
    cutilSafeCall(hipMalloc((void**)&neighborpymzGPU,numNeighbors*sizeof(int)));
    cutilSafeCall(hipMalloc((void**)&neighbormypzGPU,numNeighbors*sizeof(int)));
    cutilSafeCall(hipMalloc((void**)&neighbormymzGPU,numNeighbors*sizeof(int)));
    cutilSafeCall(hipMalloc((void**)&neighborpxpypzGPU,numNeighbors*sizeof(int)));
    cutilSafeCall(hipMalloc((void**)&neighborpxpymzGPU,numNeighbors*sizeof(int)));
    cutilSafeCall(hipMalloc((void**)&neighborpxmypzGPU,numNeighbors*sizeof(int)));
    cutilSafeCall(hipMalloc((void**)&neighborpxmymzGPU,numNeighbors*sizeof(int)));
    cutilSafeCall(hipMalloc((void**)&neighbormxpypzGPU,numNeighbors*sizeof(int)));
    cutilSafeCall(hipMalloc((void**)&neighbormxpymzGPU,numNeighbors*sizeof(int)));
    cutilSafeCall(hipMalloc((void**)&neighbormxmypzGPU,numNeighbors*sizeof(int)));
    cutilSafeCall(hipMalloc((void**)&neighbormxmymzGPU,numNeighbors*sizeof(int)));
    
    cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(cutoffGPU),&cutoff,sizeof(double)));
    double invcutoff = 1./cutoff;
    cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(invcutoffGPU),&invcutoff,sizeof(double)));
    invcutoff = 1./(cutoff * cutoff);
    cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(invcutoff2GPU),&invcutoff,sizeof(double)));
       
    

    int block = (numNeighbors-1)/128 + 1;
    cutilSafeCall(hipMalloc((void**)&pNeighbors,sizeof(vecinos)));
    initializeNeighborsFull<<<block,128>>>(neighbor0GPU,neighbor1GPU,neighbor2GPU,neighbor3GPU,neighbor4GPU,neighbor5GPU,
					   neighborpxpyGPU,neighborpxmyGPU,neighborpxpzGPU,neighborpxmzGPU,
					   neighbormxpyGPU,neighbormxmyGPU,neighbormxpzGPU,neighbormxmzGPU,
					   neighborpypzGPU,neighborpymzGPU,neighbormypzGPU,neighbormymzGPU,
					   neighborpxpypzGPU,neighborpxpymzGPU,neighborpxmypzGPU,neighborpxmymzGPU,
					   neighbormxpypzGPU,neighbormxpymzGPU,neighbormxmypzGPU,neighbormxmymzGPU, 
					   pNeighbors);

    initForcesNonBonded();
    //!*R Upload the particle type information to the GPU
    cutilSafeCall(hipMalloc((void**)&pt,sizeof(particle_type)));
    hipMalloc((void **)&particle_typesGPU, (nboundary+ np)*sizeof(int));
    hipMemcpy(particle_typesGPU, particle_types, (nboundary+np)*sizeof(int), hipMemcpyHostToDevice);
    initParticleTypes<<<1,1>>>(pt, particle_typesGPU, Aij_paramGPU, Bij_paramGPU);

  

  }


  initDelta();
  allocateErrorArray();

  cutilSafeCall(hipMalloc((void**)&pc,sizeof(particlesincell)));
  
  initParticlesInCell<<<1,1>>>(partincellX,partincellY,partincellZ,
			       countparticlesincellX,countparticlesincellY,countparticlesincellZ,
			       countPartInCellNonBonded,partInCellNonBonded,pc);

  double *auxDouble;
  auxDouble = new double [27*(nboundary+np)];
  for(int i=0;i<27*(nboundary+np);i++) auxDouble[i] = 0;
  cutilSafeCall(hipMemcpy(fxboundaryGPU,auxDouble,27*(nboundary+np)*sizeof(double),hipMemcpyHostToDevice));
  cutilSafeCall(hipMemcpy(fyboundaryGPU,auxDouble,27*(nboundary+np)*sizeof(double),hipMemcpyHostToDevice));
  cutilSafeCall(hipMemcpy(fzboundaryGPU,auxDouble,27*(nboundary+np)*sizeof(double),hipMemcpyHostToDevice));
  delete[] auxDouble;

  //Copy constant memory
  hipMemcpyToSymbol(HIP_SYMBOL(bondedForcesGPU),&bondedForces,sizeof(bool));

  
  cout << "CREATE BOUNDARIES GPU :         DONE" << endl; 

  return 1;
}
