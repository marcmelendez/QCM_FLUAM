#include "hip/hip_runtime.h"
// Filename: createCellsIncompressibleGPU.cu
//
// Copyright (c) 2010-2016, Florencio Balboa Usabiaga
//
// This file is part of Fluam
//
// Fluam is free software: you can redistribute it and/or modify
// it under the terms of the GNU General Public License as published by
// the Free Software Foundation, either version 3 of the License, or
// (at your option) any later version.
//
// Fluam is distributed in the hope that it will be useful,
// but WITHOUT ANY WARRANTY; without even the implied warranty of
// MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the
// GNU General Public License for more details.
//
// You should have received a copy of the GNU General Public License
// along with Fluam. If not, see <http://www.gnu.org/licenses/>.


#define GPUVARIABLES 1


bool createCellsQuasi2DGPU(){
  //Raul added. Upload saffman variables to gpu
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(saffmanCutOffWaveNumberGPU),&saffmanCutOffWaveNumber, sizeof(double)));
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(saffmanLayerWidthGPU),&saffmanLayerWidth, sizeof(double)));

  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(mxGPU),&mx,sizeof(int)));
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(myGPU),&my,sizeof(int)));
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(mzGPU),&mz,sizeof(int)));

  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(mxtGPU),&mxt,sizeof(int)));
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(mytGPU),&myt,sizeof(int)));
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(mztGPU),&mzt,sizeof(int)));

  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(ncellsGPU),&ncells,sizeof(int)));
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(ncellstGPU),&ncellst,sizeof(int)));
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(lxGPU),&lx,sizeof(double)));
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(lyGPU),&ly,sizeof(double)));
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(lzGPU),&lz,sizeof(double)));
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(dtGPU),&dt,sizeof(double)));
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(volumeGPU),&cVolume,sizeof(double)));
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(shearviscosityGPU),&shearviscosity,sizeof(double)));
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(temperatureGPU),&temperature,sizeof(double)));
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(thermostatGPU),&thermostat,sizeof(bool)));

  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(densfluidGPU),&densfluid,sizeof(double)));

  // Radius and kernel
  double GaussianVariance;
  // For 3D
  // double GaussianVariance = pow(hydrodynamicRadius / (1.0 * sqrt(3.1415926535897932385)), 2);
  // For 2D
  if(stokesLimit2D){
    GaussianVariance = pow(hydrodynamicRadius * 0.66556976637237890625, 2);
  }
  // For quasi-2D disks
  // double GaussianVariance = pow(hydrodynamicRadius * 9.0*sqrt(3.1415926535897932385)/16.0, 2);
  // For quasi-2D spheres
  if(quasi2D){
    GaussianVariance = pow(hydrodynamicRadius / sqrt(3.1415926535897932385), 2);
  }
  
  int kernelWidth = int(3.0 * hydrodynamicRadius * mx / lx) + 1;
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(GaussianVarianceGPU),&GaussianVariance,sizeof(double)));
  if (kernelWidth > mx/2){
    kernelWidth = mx/2;
  }
  cout << "kernelWidth = " << kernelWidth << endl;
  cout << "GaussianVariance = " << GaussianVariance << endl;
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(kernelWidthGPU),&kernelWidth,sizeof(int)));
  double deltaRFD = 1e-05 * hydrodynamicRadius;
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(deltaRFDGPU),&deltaRFD,sizeof(double)));
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(nDriftGPU),&nDrift,sizeof(int)));

  cutilSafeCall(hipMalloc((void**)&vxGPU,ncells*sizeof(double)));
  cutilSafeCall(hipMalloc((void**)&vyGPU,ncells*sizeof(double)));
  cutilSafeCall(hipMalloc((void**)&vzGPU,ncells*sizeof(double)));
 
  cutilSafeCall(hipMalloc((void**)&rxcellGPU,ncells*sizeof(double)));
  cutilSafeCall(hipMalloc((void**)&rycellGPU,ncells*sizeof(double)));
  cutilSafeCall(hipMalloc((void**)&rzcellGPU,ncells*sizeof(double)));

  // FACT1 for quasi-2D or stokesLimit2D
  double fact1 = sqrt(1.0 * temperature  / (shearviscosity * dt * lx * ly)) * ncells;
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(fact1GPU),&fact1,sizeof(double)));

  fact1 = lx/double(mx);
  double fact2 = ly/double(my);
  double fact3 = lz/double(mz);
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(dxGPU),&fact1,sizeof(double)));
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(dyGPU),&fact2,sizeof(double)));
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(dzGPU),&fact3,sizeof(double)));

  fact1 = double(mx)/lx;
  fact2 = double(my)/ly;
  fact3 = double(mz)/lz;
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(invdxGPU),&fact1,sizeof(double)));  
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(invdyGPU),&fact2,sizeof(double)));  
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(invdzGPU),&fact3,sizeof(double)));  
  fact1 = 1./dt;
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(invdtGPU),&fact1,sizeof(double)));
  fact1 = 1./lx;
  fact2 = 1./ly;
  fact3 = 1./lz;
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(invlxGPU),&fact1,sizeof(double)));  
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(invlyGPU),&fact2,sizeof(double)));  
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(invlzGPU),&fact3,sizeof(double)));

 
  bool auxbool = 0;
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(setparticlesGPU),&auxbool,sizeof(bool)));
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(setboundaryGPU),&auxbool,sizeof(bool)));

  long long auxulonglong = 0;
  cutilSafeCall(hipMalloc((void**)&stepGPU,sizeof(long long)));
  cutilSafeCall(hipMemcpy(stepGPU,&auxulonglong,sizeof(long long),hipMemcpyHostToDevice));

  //Factors for the update in fourier space
  cutilSafeCall(hipMalloc((void**)&gradKx,     mx*sizeof(hipfftDoubleComplex)));
  cutilSafeCall(hipMalloc((void**)&gradKy,     my*sizeof(hipfftDoubleComplex)));
  cutilSafeCall(hipMalloc((void**)&gradKz,     mz*sizeof(hipfftDoubleComplex)));
  cutilSafeCall(hipMalloc((void**)&expKx,      mx*sizeof(hipfftDoubleComplex)));
  cutilSafeCall(hipMalloc((void**)&expKy,      my*sizeof(hipfftDoubleComplex)));
  cutilSafeCall(hipMalloc((void**)&expKz,      mz*sizeof(hipfftDoubleComplex)));

  cutilSafeCall(hipMalloc((void**)&pF,sizeof(prefactorsFourier)));

  cutilSafeCall(hipMalloc((void**)&vxZ,ncells*sizeof(hipfftDoubleComplex)));
  cutilSafeCall(hipMalloc((void**)&vyZ,ncells*sizeof(hipfftDoubleComplex)));
  cutilSafeCall(hipMalloc((void**)&vzZ,ncells*sizeof(hipfftDoubleComplex))); 

  cout << "CREATE CELLS GPU :              DONE" << endl;

  return 1;
}
