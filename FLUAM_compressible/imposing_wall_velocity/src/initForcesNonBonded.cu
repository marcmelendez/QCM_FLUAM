#include "hip/hip_runtime.h"
// Filename: initForcesNonBonded.cu
//
// Copyright (c) 2010-2015, Florencio Balboa Usabiaga
//
// This file is part of Fluam
//
// Fluam is free software: you can redistribute it and/or modify
// it under the terms of the GNU General Public License as published by
// the Free Software Foundation, either version 3 of the License, or
// (at your option) any later version.
//
// Fluam is distributed in the hope that it will be useful,
// but WITHOUT ANY WARRANTY; without even the implied warranty of
// MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the
// GNU General Public License for more details.
//
// You should have received a copy of the GNU General Public License
// along with Fluam. If not, see <http://www.gnu.org/licenses/>.


float functionForceNonBonded1(double r){
  float sigma, epsilon;
  sigma = 2 * lx / double(mx);
  epsilon = temperature ;
  //return -epsilon * ( r - sigma);
  /********* Changed by Adolfo *********/
  // return 48. * epsilon * (pow(sigma/r,12) - 0.5*pow(sigma/r,6))/r;
  return 0;


}
float functionForceNonBonded1(double r, double a, double b){
  //  float sigma, epsilon;
  //sigma = 2 * lx / double(mx);
  //epsilon = temperature ;
  //return -epsilon * ( r - sigma);
  return (a*pow(1.0/r,12) - b*pow(1.0/r,6))/r;
}

bool initForcesNonBonded(){
  texforceNonBonded1.normalized = true;
  texforceNonBonded1.addressMode[0] = hipAddressModeClamp;//Wrap and Clamp
  texforceNonBonded1.filterMode = hipFilterModeLinear;//hipFilterModeLinear and hipFilterModePoint

  float *h_data;

  //!*R initialization of the particle type interaction
  double cutoffnear = 1.0; 
  int ntypes = 1;
  double *Aij_param;
  double *Bij_param;
  if(true){ //HELIX: Dont use types, helices interact only when their types are different, but in the same way
    Aij_param = new double;
    Bij_param = new double;
    //    float sigma, epsilon;
    //sigma = 2 * lx / float(mx); //READ FROM FILE
    //epsilon = temperature;
    *Aij_param = 0.0;//48.0f * pow(sigma,12)*epsilon;
    *Bij_param = 0.0;// 48.0f * pow(sigma,6)*0.5*epsilon;
  }
  else{    
     ifstream in("LJ.in");
     in>>ntypes;
     Aij_param = new double[ntypes*ntypes];
     Bij_param = new double[ntypes*ntypes];
     for(int i=0; i<ntypes; i++)for(int j=0; j<ntypes; j++)
				  in>>Aij_param[i+ntypes*j];
     
     
     for(int i=0; i<ntypes; i++)for(int j=0; j<ntypes; j++)
				  in>>Bij_param[i+ntypes*j];
     in>>cutoffnear;
   }
      
  //!*R Upload all the information to the GPU
  hipMalloc((void **)&Aij_paramGPU, ntypes*ntypes*sizeof(double));
  hipMalloc((void **)&Bij_paramGPU, ntypes*ntypes*sizeof(double));

  hipMemcpy(Aij_paramGPU, Aij_param, ntypes*ntypes*sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(Bij_paramGPU, Bij_param, ntypes*ntypes*sizeof(double), hipMemcpyHostToDevice);
  
  //HELIX: be carefull with this
  for(int i=1; i<np; i++){
    if(particle_types[i]!=particle_types[i-1]) ntypes++;
  }
  hipMemcpyToSymbol(HIP_SYMBOL(ntypesGPU), &ntypes, sizeof(int));
  hipMemcpyToSymbol(HIP_SYMBOL(cutoffnearGPU), &cutoffnear, sizeof(double));
  



  int size = 4096;
  h_data = new float[size];
  float r, dr;
  float cutoff2 = cutoff * cutoff;
  dr = cutoff2/float(size);
  r = 0.5 * dr;
  for(int i=0;i<size;i++){
    h_data[i] = functionForceNonBonded1(sqrt(r))/sqrt(r);
    r += dr;
  }
  h_data[size-1] = 0.;
  h_data[0] = 0.;
  hipChannelFormatDesc channelDesc = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);
  cutilSafeCall( hipMallocArray( &forceNonBonded1, &channelDesc, size, 1 )); 
  cutilSafeCall( hipMemcpyToArray( forceNonBonded1, 0, 0, h_data, size*sizeof(float), hipMemcpyHostToDevice));
  cutilSafeCall( hipBindTextureToArray( texforceNonBonded1, forceNonBonded1, channelDesc));

  /*
  r = 0.5 * dr;
    for(int i=0;i<size;i++){
    cout << r << " " << h_data[i] << endl;
    r += dr;
    }
    exit(0);
*/
  cout << "INIT FORCE NON-BONDED 1 COMPLETED" << endl;
  delete[] h_data;
  //!*R clean up
  delete[] Aij_param;
  delete[] Bij_param;
return 1;
}
